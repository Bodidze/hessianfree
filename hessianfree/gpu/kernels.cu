
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void sum_axis(float *A, float *out, const int axis, const int a0, 
                         const int a1, const int increment)
{
    // sum matrix A over the specified axis
    // TODO: replace this with a reduction kernel
    
	int a_i = blockDim.x*blockIdx.x + threadIdx.x;
	int start = 0;
	int stop = 0;
	int step = 0;
	if (axis == 0)
	{
	    if (a_i >= a1)
	       return;
		start = a_i;
		stop = a0*a1;
		step = a1;
	}
	else
	{
	    if (a_i >= a0)
	       return;
		start = a_i*a1;
		stop = start + a1;
		step = 1;
	}

	float sum = 0;
	for (int i = start; i < stop; i += step)
		sum += A[i];
	
	if (increment)
	   out[a_i] += sum;
	else
	   out[a_i] = sum;
}


__global__ void sum_cols(float *A, float *out, const int increment,
                         const int a0, const int a1)
{
    const int t_i = threadIdx.y;
    const int t_j = threadIdx.x;
    const int dim_i = blockDim.y;
    const int dim_j = blockDim.x;
    const int col = dim_j*blockIdx.x + t_j;
    const int A_offset = t_i*a1 + col;
    const int data_offset = t_i*dim_j + t_j;
    
    extern __shared__ float data[];
    
    // stage 1: loop threads across A to reduce to shared memory block
    const int step = dim_i*a1;
    const int limit = a0*a1;
    float sum = 0;
    int index = A_offset;
    for (int i=0; i < limit; i += step)
    {
        if (index < limit)
            sum += A[index];
        index += step;
    }
    data[data_offset] = sum;
    
    // stage 2: reduction within block
    // note: assumes that dim_i is divisible by 2
    for (int s=dim_i/2; s > 0; s>>=1)
    {
        __syncthreads();
        
        /*
        if (t_i == 0 && t_j == 0)
        {
            printf("data: ");
            for (int i=0; i < blockDim.x*blockDim.y; i++)
                printf("%f ", data[i]);
            printf("\n");
        }
        */
        
        if (t_i < s)
            data[data_offset] += data[data_offset + s*dim_j];
    }
    
    if (t_i == 0)
    {
        if (increment)
            out[col] += data[t_j];
        else
            out[col] = data[t_j];
    }
    
} 


__global__ void iadd(float *A, float *v, const int a0, const int a1)
{
    // in-place addition with broadcasting along first axis
    // (adding vector v to matrix A)
    
    const int row = blockDim.y*blockIdx.y + threadIdx.y;
    const int col = blockDim.x*blockIdx.x + threadIdx.x;
    
    if (row >= a0 || col >= a1)
        return;
    
    // load the appropriate part of v for this block into shared memory
    // TODO: check if this shared memory is faster than just L1 caching   
    __shared__ float v_share[32];
        
    if (threadIdx.y == 0)
        v_share[threadIdx.x] = v[col];
    
    __syncthreads();
    
    // add v to A
    A[row*a1 + col] += v_share[threadIdx.x];
    
}

